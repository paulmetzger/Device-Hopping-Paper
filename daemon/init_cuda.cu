//
// Created by paul on 01/07/2020.
//


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(float* buffer) {
    buffer[threadIdx.x] = threadIdx.x;
}

void init_cuda() {
    hipDeviceSynchronize();
#if defined(DEBUG)
    std::cout << "Initialising CUDA..." << std::endl;
#endif
    float* dummy_buffer;
    hipMalloc(&dummy_buffer, 100);
    kernel<<<100, 1>>>(dummy_buffer);
    hipDeviceSynchronize();
    hipFree(dummy_buffer);
#if defined(DEBUG)
    std::cout << "Done initialising CUDA..." << std::endl;
#endif
}